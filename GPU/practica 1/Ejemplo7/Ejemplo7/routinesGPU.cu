#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <hip/hip_runtime.h>
#include <inttypes.h>

#include "routinesGPU.h"


#define PI 3.141593
#define BLOCK_SIZE 16


__global__ void noiseReduction(uint8_t *im,float *NR,int height, int width){
	int i = threadIdx.x + blockDim.x * blockIdx.x;
	int j = threadIdx.y + blockDim.y * blockIdx.y;

	if(i >= 2 && j >= 2 && i < (height - 2) && j < (width - 2)){
					// Noise reduction
			NR[i*width+j] =
				 (2.0*im[(i-2)*width+(j-2)] +  4.0*im[(i-2)*width+(j-1)] +  5.0*im[(i-2)*width+(j)] +  4.0*im[(i-2)*width+(j+1)] + 2.0*im[(i-2)*width+(j+2)]
				+ 4.0*im[(i-1)*width+(j-2)] +  9.0*im[(i-1)*width+(j-1)] + 12.0*im[(i-1)*width+(j)] +  9.0*im[(i-1)*width+(j+1)] + 4.0*im[(i-1)*width+(j+2)]
				+ 5.0*im[(i  )*width+(j-2)] + 12.0*im[(i  )*width+(j-1)] + 15.0*im[(i  )*width+(j)] + 12.0*im[(i  )*width+(j+1)] + 5.0*im[(i  )*width+(j+2)]
				+ 4.0*im[(i+1)*width+(j-2)] +  9.0*im[(i+1)*width+(j-1)] + 12.0*im[(i+1)*width+(j)] +  9.0*im[(i+1)*width+(j+1)] + 4.0*im[(i+1)*width+(j+2)]
				+ 2.0*im[(i+2)*width+(j-2)] +  4.0*im[(i+2)*width+(j-1)] +  5.0*im[(i+2)*width+(j)] +  4.0*im[(i+2)*width+(j+1)] + 2.0*im[(i+2)*width+(j+2)])
				/159.0;
	} 
}

__global__ void gradienteImagen(float *NR,float *G, float *phi, int height, int width){
	
	int i = threadIdx.x + blockDim.x * blockIdx.x;
	int j = threadIdx.y + blockDim.y * blockIdx.y;
	float Gy, Gx, phi_thread;

	if(i >= 2 && j >= 2 && i < (height - 2) && (j < width - 2)){
		Gx = 
				 (1.0*NR[(i-2)*width+(j-2)] +  2.0*NR[(i-2)*width+(j-1)] +  (-2.0)*NR[(i-2)*width+(j+1)] + (-1.0)*NR[(i-2)*width+(j+2)]
				+ 4.0*NR[(i-1)*width+(j-2)] +  8.0*NR[(i-1)*width+(j-1)] +  (-8.0)*NR[(i-1)*width+(j+1)] + (-4.0)*NR[(i-1)*width+(j+2)]
				+ 6.0*NR[(i  )*width+(j-2)] + 12.0*NR[(i  )*width+(j-1)] + (-12.0)*NR[(i  )*width+(j+1)] + (-6.0)*NR[(i  )*width+(j+2)]
				+ 4.0*NR[(i+1)*width+(j-2)] +  8.0*NR[(i+1)*width+(j-1)] +  (-8.0)*NR[(i+1)*width+(j+1)] + (-4.0)*NR[(i+1)*width+(j+2)]
				+ 1.0*NR[(i+2)*width+(j-2)] +  2.0*NR[(i+2)*width+(j-1)] +  (-2.0)*NR[(i+2)*width+(j+1)] + (-1.0)*NR[(i+2)*width+(j+2)]);


		Gy = 
				 ((-1.0)*NR[(i-2)*width+(j-2)] + (-4.0)*NR[(i-2)*width+(j-1)] +  (-6.0)*NR[(i-2)*width+(j)] + (-4.0)*NR[(i-2)*width+(j+1)] + (-1.0)*NR[(i-2)*width+(j+2)]
				+ (-2.0)*NR[(i-1)*width+(j-2)] + (-8.0)*NR[(i-1)*width+(j-1)] + (-12.0)*NR[(i-1)*width+(j)] + (-8.0)*NR[(i-1)*width+(j+1)] + (-2.0)*NR[(i-1)*width+(j+2)]
				+    2.0*NR[(i+1)*width+(j-2)] +    8.0*NR[(i+1)*width+(j-1)] +    12.0*NR[(i+1)*width+(j)] +    8.0*NR[(i+1)*width+(j+1)] +    2.0*NR[(i+1)*width+(j+2)]
				+    1.0*NR[(i+2)*width+(j-2)] +    4.0*NR[(i+2)*width+(j-1)] +     6.0*NR[(i+2)*width+(j)] +    4.0*NR[(i+2)*width+(j+1)] +    1.0*NR[(i+2)*width+(j+2)]);
		
		G[i*width+j] = sqrtf((Gx*Gx)+(Gy*Gy));	//G = √Gx²+Gy²
		phi_thread = atan2f(fabs(Gy),fabs(Gx));

		if(fabs(phi_thread)<=PI/8 ) phi[i*width+j] = 0;
		else if (fabs(phi_thread)<= 3*(PI/8)) phi[i*width+j] = 45;
		else if (fabs(phi_thread) <= 5*(PI/8)) phi[i*width+j] = 90;
		else if (fabs(phi_thread) <= 7*(PI/8)) phi[i*width+j] = 135;
		else phi[i*width+j] = 0;
	}
}

__global__ void edge(float *G, float *phi,uint8_t *image_out,int height, int width){
	int i = threadIdx.x + blockDim.x * blockIdx.x;
	int j = threadIdx.y + blockDim.y * blockIdx.y;
	int ii, jj;
	float lowthres, hithres;
	uint8_t pedge = 0;

	if(i >= 3 && j >= 3 && i < (height - 3) && (j < width - 3)){
		if(phi[i*width+j] == 0){
				if(G[i*width+j]>G[i*width+j+1] && G[i*width+j]>G[i*width+j-1]) //edge is in N-S
					pedge = 1;

			} else if(phi[i*width+j] == 45) {
				if(G[i*width+j]>G[(i+1)*width+j+1] && G[i*width+j]>G[(i-1)*width+j-1]) // edge is in NW-SE
					pedge = 1;

			} else if(phi[i*width+j] == 90) {	
				if(G[i*width+j]>G[(i+1)*width+j] && G[i*width+j]>G[(i-1)*width+j]) //edge is in E-W
					pedge = 1;

			} else if(phi[i*width+j] == 135) {
				if(G[i*width+j]>G[(i+1)*width+j-1] && G[i*width+j]>G[(i-1)*width+j+1]) // edge is in NE-SW
					pedge = 1;
			}
	
		lowthres = 1000.0f/2;
		hithres  = 2*(1000.0f);

		image_out[i*width+j] = 0;
		if(G[i*width+j]>hithres && pedge)
			image_out[i*width+j] = 255;
		else if(pedge && G[i*width+j]>=lowthres && G[i*width+j]<hithres)
			// check neighbours 3x3
			for (ii=-1;ii<=1; ii++)
				for (jj=-1;jj<=1; jj++)
					if (G[(i+ii)*width+j+jj]>hithres){
						image_out[i*width+j] = 255;
					//	ii = 2;
						//jj=2;
					}
	}
	
	
}
__global__
void houghtransform(uint8_t *im, uint32_t *accumulators, float *sin_table, float *cos_table, int height, int width)
{

	int i = threadIdx.x + blockDim.x * blockIdx.x;
	int j = threadIdx.y + blockDim.y * blockIdx.y; 
	int theta;

	float hough_h = ((sqrt(2.0) * (float)(height>width?height:width)) / 2.0);

	

	float center_x = width/2.0; 
	float center_y = height/2.0;
	if(i >= 0 && j >= 0 && i < height && j < width){ //Quitar los mayor = que 0
		if( im[ (i*width) + j] > 250 ) // Pixel is edge  
		{  
			for(theta=0;theta<180;theta++)  
			{  
				float rho = ( ((float)j - center_x) * cos_table[theta]) + (((float)i - center_y) * sin_table[theta]);
				atomicAdd(&accumulators[(int)((round(rho + hough_h) * 180.0)) + theta],1);


			} 
		} 
	}	 
}

void getlines(int threshold, uint32_t *accumulators, int accu_width, int accu_height, int width, int height, 
	float *sin_table, float *cos_table,
	int *x1_lines, int *y1_lines, int *x2_lines, int *y2_lines, int *lines)
{
	int rho, theta;
	uint32_t max;
	for(rho=0;rho<accu_height;rho++)
	{

		for(theta=0;theta<accu_width;theta++)  
		{  
			
			if(accumulators[(rho*accu_width) + theta] >= threshold)  
			{  
				//Is this point a local maxima (9x9)  
				max = accumulators[(rho*accu_width) + theta]; 
				
				for(int ii=-4;ii<=4;ii++)  
				{  
					for(int jj=-4;jj<=4;jj++)  
					{  
						if( (ii+rho>=0 && ii+rho<accu_height) && (jj+theta>=0 && jj+theta<accu_width) )  
						{  
							if( accumulators[((rho+ii)*accu_width) + (theta+jj)] > max )  
							{
								max = accumulators[((rho+ii)*accu_width) + (theta+jj)];
							}  
						}  
					}  
				}  

				if(max == accumulators[(rho*accu_width) + theta]) //local maxima
				{
					int x1, y1, x2, y2;  
					x1 = y1 = x2 = y2 = 0;  

					if(theta >= 45 && theta <= 135)  
					{
						if (theta>90) {
							//y = (r - x cos(t)) / sin(t)  
							x1 = width/2;  
							y1 = ((float)(rho-(accu_height/2)) - ((x1 - (width/2) ) * cos_table[theta])) / sin_table[theta] + (height / 2);
							x2 = width;  
							y2 = ((float)(rho-(accu_height/2)) - ((x2 - (width/2) ) * cos_table[theta])) / sin_table[theta] + (height / 2);  
						} else {
							//y = (r - x cos(t)) / sin(t)  
							x1 = 0;  
							y1 = ((float)(rho-(accu_height/2)) - ((x1 - (width/2) ) * cos_table[theta])) / sin_table[theta] + (height / 2);
							x2 = width*2/5;  
							y2 = ((float)(rho-(accu_height/2)) - ((x2 - (width/2) ) * cos_table[theta])) / sin_table[theta] + (height / 2); 
						}
					} else {
						//x = (r - y sin(t)) / cos(t);  
						y1 = 0;  
						x1 = ((float)(rho-(accu_height/2)) - ((y1 - (height/2) ) * sin_table[theta])) / cos_table[theta] + (width / 2);  
						y2 = height;  
						x2 = ((float)(rho-(accu_height/2)) - ((y2 - (height/2) ) * sin_table[theta])) / cos_table[theta] + (width / 2);  
					}
					x1_lines[*lines] = x1;
					y1_lines[*lines] = y1;
					x2_lines[*lines] = x2;
					y2_lines[*lines] = y2;
					(*lines)++;
				}
			}
		}
	}
}



void line_asist_GPU(uint8_t *im, int height, int width,
	uint8_t *imEdge, float *NR, float *G, float *phi, float *Gx, float *Gy, uint8_t *pedge,
	float *sin_table, float *cos_table, 
	uint32_t *accum, int accu_height, int accu_width,
	int *x1, int *x2, int *y1, int *y2, int *nlines)
{
	//printf("%d",height);
	/* To do */

	
	uint8_t *imGPU;
	float *NRGPU;
	float *GGPU;
	float *phiGPU;
	uint8_t *im_outGPU;
	uint32_t *accumGPU;
	int threshold;
	float *sin_tableGPU;
	float *cos_tableGPU;
	
	
	hipMalloc((void**)&imGPU, sizeof(uint8_t)*height*width);
	hipMalloc((void**)&NRGPU, sizeof(float)*height*width);
	hipMalloc((void**)&GGPU, sizeof(float)*height*width);
	hipMalloc((void**)&phiGPU, sizeof(float)*height*width);;
	hipMalloc((void**)&im_outGPU, sizeof(uint8_t)*height*width);

	dim3 threadsPerBlock(BLOCK_SIZE,BLOCK_SIZE);	
	dim3 numBlocks((height/BLOCK_SIZE + 1),(width/BLOCK_SIZE)+1);
	
	hipMemcpy(imGPU,im, sizeof(uint8_t)*height*width, hipMemcpyHostToDevice);
	noiseReduction<<<numBlocks,threadsPerBlock>>>(imGPU, NRGPU, height, width);	

	gradienteImagen<<<numBlocks,threadsPerBlock>>>(NRGPU, GGPU, phiGPU,height, width);

	edge<<<numBlocks,threadsPerBlock>>>(GGPU, phiGPU,im_outGPU, height, width);


	//hipMemcpy(pedgeGPU,im, sizeof(uint8_t)*height*width, hipMemcpyHostToDevice);

	hipMalloc((void**)&accumGPU, accu_width*accu_height*sizeof(uint32_t));
	hipMalloc((void**)&sin_tableGPU, sizeof(float)*180);
	hipMalloc((void**)&cos_tableGPU, sizeof(float)*180);
	
	hipMemcpy(sin_tableGPU,sin_table, sizeof(float)*180, hipMemcpyHostToDevice);
	hipMemcpy(cos_tableGPU,cos_table, sizeof(float)*180, hipMemcpyHostToDevice);

	hipMemset(accumGPU, 0, (sizeof(uint32_t) * accu_width * accu_height));
	houghtransform<<<numBlocks,threadsPerBlock>>>(im_outGPU,accumGPU, sin_tableGPU, cos_tableGPU,height, width);
	//Pasarselo al accum
	hipMemcpy(accum, accumGPU, accu_width*accu_height*sizeof(uint32_t), hipMemcpyDeviceToHost);

	if (width>height) threshold = width/6;
	else threshold = height/6;

	
	getlines(threshold, accum, accu_width, accu_height, width, height, 
		sin_table, cos_table,
		x1, y1, x2, y2, nlines);


	hipFree(imGPU);	
	hipFree(NR);	
	hipFree(GGPU);
	hipFree(phiGPU);
	hipFree(im_outGPU);
	hipFree(accumGPU);
	hipFree(sin_tableGPU);
	hipFree(cos_tableGPU);
			
			
	
}
