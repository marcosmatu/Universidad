#include <stdio.h>
#include "hipblas.h"
#include "matrix_mul.h"

// Host multiplication function
// Compute C = A * B
// hA is the height of A
// wA is the width of A
// wB is the width of B

extern "C"
void Mul(float* A, float* B, int hA, int wA, int wB,
	float* C)
{
	int size;

	// Load A and B to the device
	float* Ad;
	size = hA * wA * sizeof(float);
	hipMalloc((void**)&Ad, size);
	hipMemcpy(...);
	float* Bd;
	size = wA * wB * sizeof(float);
	hipMalloc((void**)&Bd, size);
	hipMemcpy(...);

	// Allocate C on the device
	float* Cd;
	size = hA * wB * sizeof(float);
	hipMalloc((void**)&Cd, size);

	// Compute the execution configuration
	hipblasSgemm( ...
		...,				/* [m] */ 
		..,				/* [n] */  
		..,				/* [k] */ 
		1,				/* alfa */ 
		..., ...,			/* A[m][k], num columnas (lda) */ 
		..., ...,			/* B[k][n], num columnas (ldb) */
		0,				/* beta */
		..., ...			/* C[m][n], num columnas (ldc) */
	);

	// Read C from the device
	hipMemcpy(C, Cd, ..., ...);

	// Free device memory
	hipFree(Ad);
	hipFree(Bd);
	hipFree(Cd);
}
